#include <ScalarMulRunner.cuh>

#include <ScalarMul.cuh>
#include <KernelMul.cuh>
#include <CommonKernels.cuh>
#include <cassert>

float ScalarMulTwoReductions(int numElements, float *vector1, float *vector2, int blockSize) {

  float *d_vector_1 = nullptr;
  float *d_vector_2 = nullptr;
  float *d_res = nullptr;

  size_t blocks_count = (numElements + blockSize - 1) / blockSize;

  hipMalloc(&d_vector_1, numElements * sizeof(float));
  hipMalloc(&d_vector_2, numElements * sizeof(float));
  hipMalloc(&d_res, sizeof(float));
  hipMemset(d_res, 0.f, sizeof(float));

  assert(d_vector_1 != nullptr);
  assert(d_res != nullptr);
  assert(d_vector_2 != nullptr);

  hipMemcpy(d_vector_1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vector_2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  ScalarMulBlock<<<blocks_count, blockSize, sizeof(float) * blockSize>>>(numElements,
                                                                         d_vector_1,
                                                                         d_vector_2,
                                                                         d_res);

  float scalar_product = 0.f;
  hipMemcpy(&scalar_product, d_res, sizeof(float), hipMemcpyDeviceToHost);

  float total_time = 0;

  float seq_scalar_product = 0.f;

  for (size_t i = 0; i < numElements; ++i) {
    seq_scalar_product += vector1[i] * vector2[i];
  }

  hipFree(d_vector_1);
  hipFree(d_vector_2);
  hipFree(d_res);

  return scalar_product;
}

float ScalarMulSumPlusReduction(int numElements, float *vector1, float *vector2, int blockSize) {

  float *d_vector_1 = nullptr;
  float *d_vector_2 = nullptr;
  float *d_product = nullptr;
  float *d_res = nullptr;

  size_t blocks_count = (numElements + blockSize - 1) / blockSize;

  hipMalloc(&d_vector_1, numElements * sizeof(float));
  hipMalloc(&d_vector_2, numElements * sizeof(float));
  hipMalloc(&d_product, numElements * sizeof(float));
  hipMalloc(&d_res, sizeof(float));
  hipMemset(d_res, 0.f, sizeof(float));

  assert(d_vector_1 != nullptr);
  assert(d_res != nullptr);
  assert(d_vector_2 != nullptr);
  assert(d_product != nullptr);

  hipMemcpy(d_vector_1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vector_2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  KernelMul<<<blocks_count, blockSize>>>(numElements, d_vector_1, d_vector_2, d_product);

  hipDeviceSynchronize();

  Reduce<<<blocks_count, blockSize, sizeof(float) * blockSize>>>(numElements,
                                                                 d_product,
                                                                 d_res);

  float scalar_product = 0.f;

  hipMemcpy(&scalar_product, d_res, sizeof(float), hipMemcpyDeviceToHost);

  float total_time = 0;

  float seq_scalar_product = 0.f;

  for (size_t i = 0; i < numElements; ++i) {
    seq_scalar_product += vector1[i] * vector2[i];
  }

  hipFree(d_vector_1);
  hipFree(d_vector_2);
  hipFree(d_res);
  hipFree(d_product);

  return scalar_product;
}
